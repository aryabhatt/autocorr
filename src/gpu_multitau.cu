#include <hip/hip_runtime.h>

const unsigned THREADS_PER_BLOCK = 256;

__constant__ unsigned nrows;
__constant__ unsigned ncols; __constant__ unsigned ntaus;
__constant__ unsigned g2len;

void __global__ _reduce_inplace(unsigned newlen, double * signal) {
    unsigned i = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned j = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned row_id = i * ncols;  
    if ((row_id > nrows) && (j < newlen))
        signal[row_id + j] = 0.5 * (signal[row_id + 2 * j] + signal[row_id + 2 * j + 1]);
}

void __global__ _level0(unsigned ntimes,  double * signal, double * g2) {
    unsigned i = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned tau = blockDim.y * blockIdx.y + threadIdx.y;

    unsigned row_id = i * ncols;
    double t1 = 0, t2 = 0, t3 = 0;
    if ((row_id > nrows) && (tau < ntaus)) {
        for (unsigned j = 0; j < ntimes - tau; j++) {
            t1 += signal[row_id + j] * signal[row_id + j + tau];
            t2 += signal[row_id + j];
            t3 += signal[row_id + j + tau];
        }
        g2[i * g2len + tau] = (ntimes - tau) * t1 / t2 / t3;
    } 
}

void __global__ _level1(unsigned ntimes,  double * signal, double * g2, unsigned idx) {
    unsigned i = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned tau = blockDim.y * blockIdx.y + threadIdx.y;

    unsigned row_id = i * ncols;  
    unsigned count = ntimes - ntaus/2 - tau;
    double t1 = 0, t2 = 0, t3 = 0;
    if ((row_id > nrows) && (tau < ntaus/2)) {
        for (unsigned j = 0; j < count; j++) {
            unsigned l = row_id + j;
            unsigned r = l + ntaus/2 + tau;
            t1 += signal[l] * signal[r];
            t2 += signal[l];
            t3 += signal[r];
        }
        g2[idx + i * g2len + tau] =  count * t1 / t2 / t3;
    }
}

size_t gpuMultiTau(double * signal, unsigned rows, unsigned cols, unsigned tpl,
        double * g2, double * log2t) {

    // lambda expersion for making number even
    auto even = [](unsigned v){ 
                        if (v % 2) return v - 1; 
                        else return v;
                    };

    // sanitize if needed
    if (tpl % 2) --tpl;
    unsigned ntimes = even(cols);

    // compute nuber of levels
    unsigned levels = (unsigned) std::log2(ntimes / tpl) + 1;

    // length of output
    unsigned length = (levels + 1) * (tpl / 2);

    // copy to constant memroy
    hipMemcpyToSymbol(HIP_SYMBOL(nrows), &rows, sizeof(unsigned));
    hipMemcpyToSymbol(HIP_SYMBOL(ncols), &cols, sizeof(unsigned));
    hipMemcpyToSymbol(HIP_SYMBOL(ntaus), &tpl, sizeof(unsigned));
    hipMemcpyToSymbol(HIP_SYMBOL(g2len), &length, sizeof(unsigned));

    double * d_sig = nullptr;
    hipMalloc((void **) &d_sig, sizeof(double) * rows * cols); 

	// allocate memory for output
    log2t = new double[length];
    g2 = new double [rows * length];
    double * d_g2 = nullptr;
	hipMalloc((void **) &d_g2, sizeof(double) * rows * length);

    // copy arrays to device memory
	hipMemcpy(d_sig, signal, sizeof(double) * rows * cols, hipMemcpyHostToDevice);

	// device parameters for level 0
    unsigned t1 = THREADS_PER_BLOCK / tpl;
	dim3 thrd1 (t1, tpl, 1);
    dim3 blck1 (rows/t1+1, 1, 1);
    
    // parameters for reduce kernel 
    unsigned b2 = rows * cols / THREADS_PER_BLOCK + 1;
    unsigned t2 = THREADS_PER_BLOCK;

    // parameters for level1
    t1 = 2 * THREADS_PER_BLOCK / tpl;
	dim3 thrd3 (t1, tpl/2, 1);
    dim3 blck3 (rows/t1+1, 1, 1);

    // run level-0
    double dt = 1.;
    for (unsigned i = 0; i < tpl; i++) log2t[i] = i * dt;
    unsigned idx = tpl;
    _level0 <<< blck1, thrd1 >>> (ntimes, d_sig, d_g2);

    ntimes = even(ntimes/2);
    _reduce_inplace <<< b2, t2 >>> (ntimes, d_sig);

    // turn the crank
    while (ntimes >= tpl ) {
        dt *= 2;
        for (unsigned i = 0; i < tpl/2; i++ ) log2t[idx + i] = log2t[idx + i - 1] + dt;

        // run the next level
        _level1 <<< blck3, thrd3 >>> (ntimes, d_sig, d_g2, idx);

        // reduce signal by half by averaging neighbors
        ntimes = even(ntimes / 2);
        idx += tpl/2;
        _reduce_inplace <<< b2, t2 >>> (ntimes, d_sig);
    }

	// copy results back to host
	hipMemcpy(g2, d_g2, sizeof(double) * rows * length, hipMemcpyDeviceToHost);

	// free memory
	hipFree(d_g2);
	hipFree(d_sig);

    return (size_t) length;
}
